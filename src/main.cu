#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdlib>
#include <cmath>
#include <cstring>
#include <random>
#include <vector>
#include <algorithm>
#include <hip/hip_fp16.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "utils.h"
#include "init_curand_states.h"
#include "flash_forward.h"


void verify(half* O, half* O_host, const int batch_size, const int n_heads, const int seq_len, const int head_dim, float range_of_error);

void attention_forward_cpu(const half* Q, const half* K, const half* V, float softmax_scale, const int batch_size, const int n_heads, const int seq_len, 
    const int head_dim, half* output, const bool use_causal_mask = false, int window_size = -1, const float* alibi_slopes = nullptr);

int main(){
    int  batch_size       = 2;
    int  n_heads          = 8;
    int  seq_len          = 2048;
    int  head_dim         = 64;

    bool dropout          = false;
    bool causal_mask      = false;
    bool window_attention = false;
    bool alibi            = false;
    float dropout_prob    = 0.0f;
    int window_size       = -1;

    hiprandStatePhilox4_32_10_t* d_states;

    float *alibi_slopes = nullptr;
    float *alibi_slopes_device = nullptr;
    if (alibi) {
        alibi_slopes = (float*)malloc(n_heads*sizeof(float));
        for (int i = 0; i < n_heads; i++){
            alibi_slopes[i] = -std::pow(2, -8.0 / n_heads * (i + 1));
        }
        hipMalloc((void**)&alibi_slopes_device, n_heads*sizeof(float));
        hipMemcpy(alibi_slopes_device, alibi_slopes, n_heads*sizeof(float),hipMemcpyHostToDevice);
    }

    if (window_attention) {
        window_size = 128;
    }
      
    float *Q = (float*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(float));
    float *K = (float*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(float));
    float *V = (float*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(float));

    half *Q_half = (half*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(half));
    half *K_half = (half*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(half));
    half *V_half = (half*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(half));
    half *O_half = (half*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(half));
    half *O_host = (half*)malloc(batch_size*n_heads*seq_len*head_dim*sizeof(half));

    half  *Q_device,*K_device,*V_device, *O_device;
    hipMalloc((void**)&Q_device, batch_size*n_heads*seq_len*head_dim*sizeof(half));
    hipMalloc((void**)&K_device, batch_size*n_heads*seq_len*head_dim*sizeof(half));
    hipMalloc((void**)&V_device, batch_size*n_heads*seq_len*head_dim*sizeof(half));
    hipMalloc((void**)&O_device, batch_size*n_heads*seq_len*head_dim*sizeof(half));

    float* O_tmp; float* L; float* M;

    std::default_random_engine generator(26);
    std::uniform_real_distribution<float> distribution(0.0f, 10.0f);
    for(int i = 0; i < batch_size*n_heads*seq_len*head_dim; i++)
    {
        Q[i] = distribution(generator);
        K[i] = distribution(generator);
        V[i] = distribution(generator);

        Q_half[i] = __float2half(Q[i]);
        K_half[i] = __float2half(K[i]);
        V_half[i] = __float2half(V[i]);
        O_half[i] = 0;
    }

    hipMemcpy(Q_device, Q_half, batch_size*n_heads*seq_len*head_dim*sizeof(half),hipMemcpyHostToDevice);
    hipMemcpy(K_device, K_half, batch_size*n_heads*seq_len*head_dim*sizeof(half),hipMemcpyHostToDevice);
    hipMemcpy(V_device, V_half, batch_size*n_heads*seq_len*head_dim*sizeof(half),hipMemcpyHostToDevice);

    if (dropout) {
        // 分配状态内存
        int num_blocks = ceil((float)seq_len / 128) * n_heads * batch_size * 256;
        hipMalloc(&d_states, num_blocks * sizeof(hiprandStatePhilox4_32_10_t));

        // 初始化状态
        dim3 grid((num_blocks + 255)/256, 1, 1);
        int seed = 48;
        init_curand_states<<<grid, 256>>>(d_states, seed, num_blocks);
    }

    // GPU端计算结果
    run_flash_attention(batch_size, n_heads, seq_len, head_dim, Q_device, K_device, V_device, O_device, nullptr, nullptr, dropout, causal_mask, window_attention, alibi, window_size, alibi_slopes_device, dropout_prob, d_states);

    hipMemcpy(O_host, O_device, batch_size*n_heads*seq_len*head_dim*sizeof(half), hipMemcpyDeviceToHost);
    // 检验结果正确性
    if(!dropout){
        printf("Verify the result of kernel function\n");
        // CPU端计算正确结果
        attention_forward_cpu(Q_half, K_half, V_half, 1.0 / sqrt(head_dim), batch_size, n_heads, seq_len, head_dim, O_half, causal_mask, window_size, alibi_slopes);
        verify(O_half, O_host, batch_size, n_heads, seq_len, head_dim, 0.06);
    }

    // 释放显存
    hipFree(O_device);
    hipFree(Q_device);
    hipFree(K_device);
    hipFree(V_device);
    hipFree(d_states);
    
    hipFree(L);
    hipFree(M);
    hipFree(O_tmp);
    
    // 释放内存
    free(Q);
    free(K);
    free(V);
    free(O_half);
    free(O_host);
    free(Q_half);
    free(K_half);
    free(V_half);
    
    return 0;
}

void verify(
    half* O, 
    half* O_host,
    const int batch_size,
    const int n_heads,
    const int seq_len,
    const int head_dim,
    float range_of_error)
{
    int error=0;
    printf("===================start verify===================\n");
    for(int i=0;i<batch_size*n_heads*seq_len*head_dim;i++)
    {
        float device_out = __float2half(O_host[i]);
        float host_out = __float2half(O[i]);
        if((fabs(device_out - host_out))/host_out > range_of_error || std::isnan(device_out) || std::isinf(device_out))
        {
            printf("error, postion:%d, gpuvalue:%f, cpuvalue:%f\n", i, device_out, host_out);
            error++;
            break;
        }        
    }
    printf("==================finish,error:%d==================\n",error);
}

void attention_forward_cpu(
    const half* Q,
    const half* K,
    const half* V,
    float softmax_scale,
    const int batch_size,
    const int n_heads,
    const int seq_len,
    const int head_dim,
    half* output,
    const bool use_causal_mask,
    int window_size,
    const float* alibi_slopes)
{
    const int head_size = seq_len * head_dim;
    const int seq_sq = seq_len * seq_len;

    // 临时存储注意力分数
    float* scores = new float[seq_sq];

    for (int b = 0; b < batch_size; ++b) {
        for (int h = 0; h < n_heads; ++h) {
            // 获取当前head的指针偏移量
            const int base_offset = b * n_heads * head_size + h * head_size;
            const half* Q_ptr = Q + base_offset;
            const half* K_ptr = K + base_offset;
            const half* V_ptr = V + base_offset;
            half* out_ptr = output + base_offset;

            // 1. 计算QK^T
            for (int i = 0; i < seq_len; ++i) {
                for (int j = 0; j < seq_len; ++j) {
                    float sum = 0.0f;
                    for (int k = 0; k < head_dim; ++k) {
                        sum += __half2float(Q_ptr[i * head_dim + k] * K_ptr[j * head_dim + k]);
                    }
                    scores[i * seq_len + j] = sum * softmax_scale;
                }
            }

            // 2. 应用ALiBi偏置
            if (alibi_slopes != nullptr) {
                const float slope = alibi_slopes[h];
                for (int i = 0; i < seq_len; ++i) {
                    for (int j = 0; j < seq_len; ++j) {
                        scores[i * seq_len + j] -= slope * std::abs(i - j);
                    }
                }
            }

            // 3. 应用注意力掩码
            if (use_causal_mask) {
                for (int i = 0; i < seq_len; ++i) {
                    for (int j = 0; j < seq_len; ++j) {
                        if (j > i) {
                            scores[i * seq_len + j] = -INFINITY;
                        }
                    }
                }
            }

            if (window_size >= 0) {
                const int w = window_size;
                for (int i = 0; i < seq_len; ++i) {
                    for (int j = 0; j < seq_len; ++j) {
                        if (std::abs(i - j) > w) {
                            scores[i * seq_len + j] = -INFINITY;
                        }
                    }
                }
            }

            // 4. Softmax计算
            for (int i = 0; i < seq_len; ++i) {
                float max_val = -INFINITY;
                float* row = scores + i * seq_len;
                
                // 计算行最大值
                for (int j = 0; j < seq_len; ++j) {
                    max_val = std::max(max_val, row[j]);
                }

                // 计算指数和
                float sum = 0.0f;
                for (int j = 0; j < seq_len; ++j) {
                    row[j] = expf(row[j] - max_val);
                    sum += row[j];
                }

                // 归一化
                for (int j = 0; j < seq_len; ++j) {
                    row[j] /= sum;
                }
            }

            // 5. 计算加权和
            for (int i = 0; i < seq_len; ++i) {
                for (int k = 0; k < head_dim; ++k) {
                    float sum = 0.0f;
                    for (int j = 0; j < seq_len; ++j) {
                        sum += __half2float(__float2half(scores[i * seq_len + j]) * V_ptr[j * head_dim + k]);
                    }
                    out_ptr[i * head_dim + k] = __float2half(sum);
                }
            }
        }
    }

    delete[] scores;
}